#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <vector>
#include <fstream>
#include <string>
#include <chrono>
#include <iomanip>
#include <stack>
#include <thread>
#include <hip/hip_runtime.h>
#include <>

#include "gpuErrchk.h"

using namespace std;


#define N_MAX 10

// Determine the validity of the board
bool boardIsValid(const int* gameBoard, const int N)
{
    for (int i = 0; i < N; i++)
        for (int j = i + 1; j < N; j++)
            if (gameBoard[i] - gameBoard[j] == i - j || gameBoard[i] - gameBoard[j] == j - i || gameBoard[i] == gameBoard[j])
                return false;
    return true;
}

// Calculate the solutions
void calculateSolutions(int N, std::vector<std::vector<int>>& solutions)
{
    int O = pow(N, N);

    int** solutionArr = nullptr;
    int no_of_sols = 0;

    auto start = std::chrono::system_clock::now();

    for (int i = 0; i < O; i++) {
        int* gameBoard = (int*)malloc(sizeof(int) * N);

        int column = i;
        for (int j = 0; j < N; j++) {
            gameBoard[j] = column % N;
            column /= N;
        }

        if (boardIsValid(gameBoard, N)) {
            no_of_sols++;
            solutionArr = (int**)realloc(solutionArr, sizeof(int*) * no_of_sols);
            solutionArr[no_of_sols - 1] = gameBoard;
        }
    }

    auto stop = std::chrono::system_clock::now();
    auto time_elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "N=" << N << " time elapsed: " << time_elapsed.count() / 1000.0 << "s\n";

    // Add the solution to the solutions array
    for (int i = 0; i < no_of_sols; i++) {
        solutions.push_back(std::vector<int>(solutionArr[i], solutionArr[i] + sizeof solutionArr[i] / sizeof solutionArr[i][0]));
        free(solutionArr[i]);
    }
    // Free memory
    free(solutionArr);
}

// Calculate all solutions given the size of the chessboard
void calculateAllSolutions(int N, bool print)
{
    std::vector<std::vector<int>> solutions;

    calculateSolutions(N, solutions);
    printf("N=%d, solutions=%d\n\n", N, int(solutions.size()));

}


int main(int argc, char** argv)
{

    for (int N = 4; N <= N_MAX; ++N)
        calculateAllSolutions(N, false);

    /*
    // Input Specific N for solution
{
    int n;
    char c;

    do
    {
        cout << "-= NQueens Puzzle Solutions\n";
        cout << "-= WARNING: N > 9 will take longer to process \n";
        cout << "-= N = 10 takes around 75 seconds on my PC \n\n";

        do
        {
            cout << "\tEnter an between 3 and 15 (not-inclusive) \n";
            cout << "\nN = ";
            cin >> n;
            if (n < 4 || n > 14)
                cout << "INVALID!\n";
        } while (n < 4 || n > 14);

        calculateAllSolutions(n, false);

        cout << "Solve another N-Queen Puzzle (Y/N) ? ";
        cin >> c;

        cout << "\n\n";
    } while (c != 'N' && c != 'n');

    system("PAUSE");
    return 0;
}
*/

}