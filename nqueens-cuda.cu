#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <chrono>
#include <iomanip>
#include <stack>
#include <thread>
#include <algorithm>
#include <sstream>
#include <cmath>

// CUDA Includes
#include <hip/hip_runtime.h>
#include <>

// Error check Helper 
#include "gpuErrchk.h"


#define N_MAX 10 // Max size of the board (10X10)
#define THREADPERBLOCK 512 // allocates 2D GPU threads (max 1024, 512 has been chosen as a middleground and it seems the most efficient)

// __device__ to indicate use on the GPU 
__device__ bool boardIsValidSoFar(int lastPlacedRow, const int* gameBoard, const int N)
{
    int lastPlacedColumn = gameBoard[lastPlacedRow];

    // Check against other queens
    for (int row = 0; row < lastPlacedRow; ++row)
    {
        if (gameBoard[row] == lastPlacedColumn) // same column, fail!
            return false;
        // check the 2 diagonals
        const auto col1 = lastPlacedColumn - (lastPlacedRow - row);
        const auto col2 = lastPlacedColumn + (lastPlacedRow - row);
        if (gameBoard[row] == col1 || gameBoard[row] == col2)
            return false;
    }
    return true;
}

//__global__ to indicate use by GPU by multiple threads
__global__ void checkQueenPos(const int N, const long long int O, const long long int offset, int* d_solutions, int* d_no_of_sols)
{
    // Column = threadIdx.x + blockIdx.x * blockDim.x
    long long int column = (long long int)(threadIdx.x + blockIdx.x * blockDim.x);
    if (column >= O)
        return;
    bool valid = true;

    // Game Board Array
    int gameBoard[N_MAX];

    // Checking Queen Positions
    for (int i = 0; i < N; i++) {
        gameBoard[i] = column % N;

        if (!boardIsValidSoFar(i, gameBoard, N)) {
            valid = false;
            break;
        }

        // divide and assign to column
        column /= N;
    }

    // If the board is valid
    if (valid) {
        const int index = atomicAdd(d_no_of_sols, 1);
        for (int i = 0; i < N; i++)
            d_solutions[N * index + i] = gameBoard[i] + 1; // Increment number of device solutions
    }
}

// Calculate the solutions to the problem
void calculateSolutions(const int N, std::vector<std::vector<int>>* solutions, int* h_no_of_sols)
{
    // h for host variables
    *h_no_of_sols = 0;
    // d for device variables
    int* d_solutions = nullptr;
    int* d_no_of_sols = nullptr;

    // For board evaluation
    const long long int O = powl(N, N);

    //Solutions Array and Number of solutions
    size_t solutions_mem = pow(N, 5) * sizeof(int*);
    hipMalloc((void**)&d_solutions, solutions_mem);
    hipMalloc((void**)&d_no_of_sols, sizeof(int));

    // copy host host number of solutions to device number of solutions
    hipMemcpy(d_no_of_sols, h_no_of_sols, sizeof(int), hipMemcpyHostToDevice);

    // Defining grid and blocks
    long long int grid = (O + THREADPERBLOCK - 1) / THREADPERBLOCK;
    int block = THREADPERBLOCK;

    for (long long int i = 0; i < 1; i++) {
        checkQueenPos << <grid, block >> > (N, O, NULL, d_solutions, d_no_of_sols); //kernel for checking the queen positions
        hipDeviceSynchronize(); // host device ensures device synchronisation
    }

    // Copy device number of solutions to host number of solutions
    hipMemcpy(h_no_of_sols, d_no_of_sols, sizeof(int), hipMemcpyDeviceToHost);
    // Free up memory of device number of solutions
    hipFree(d_no_of_sols);

    int* h_solutions = (int*)malloc(solutions_mem);
    hipMemcpy(h_solutions, d_solutions, solutions_mem, hipMemcpyDeviceToHost);
    hipFree(d_solutions);

    // Add solutions to the solutions array
    for (int i = 0; i < *h_no_of_sols; i++) {
        if (h_solutions[N * i] != NULL) {
            std::vector<int> solution = std::vector<int>();
            for (int j = 0; j < N; j++)
                solution.push_back(h_solutions[N * i + j]);
            solutions->push_back(solution);
        }
    }

    // Free memory of host solutions
    free(h_solutions);
}

void calculateAllSolutions(const int N, const bool print)
{
    std::vector<std::vector<int>> solutions = std::vector<std::vector<int>>();
    int no_of_sols = 0;

    auto startTime = std::chrono::system_clock::now();
    calculateSolutions(N, &solutions, &no_of_sols);
    auto stopTime = std::chrono::system_clock::now();

    auto timeTaken = std::chrono::duration_cast<std::chrono::microseconds>(stopTime - startTime);
    std::cout << "N=" << N << " Solution found in: " << timeTaken.count() / 1000000.0 << "s\n";
    printf("N=%d, solutions=%d\n\n", N, no_of_sols);

}

int main(int argc, char** argv)
{
    // Helper to exit on first CUDA error
    gpuErrchk(hipSetDevice(0));

    for (int N = 4; N <= N_MAX; ++N)
        calculateAllSolutions(N, false);

    /*
// Input Specific N for solution
{
    int n;
    char c;

    do
    {
        cout << "-= NQueens Puzzle Solutions\n";

        do
        {
            cout << "\tEnter an between 3 and 15 (not-inclusive) \n";
            cout << "\nN = ";
            cin >> n;
            if (n < 4 || n > 14)
                cout << "INVALID!\n";
        } while (n < 4 || n > 14);

        calculateAllSolutions(n, false);

        cout << "Solve another N-Queen Puzzle (Y/N) ? ";
        cin >> c;

        cout << "\n\n";
    } while (c != 'N' && c != 'n');

    system("PAUSE");
    return 0;
}
*/
}